
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello() {
    // 不使用 printf
}

int main() {
    hello<<<1, 1>>>();
    hipError_t err = hipDeviceSynchronize();

    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    std::cout << "CUDA Kernel executed successfully!" << std::endl;
    return 0;
}

